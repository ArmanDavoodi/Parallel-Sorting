#include "hip/hip_runtime.h"
#include "sequential/sequential_sorts.h"
#include "openmp/omp_sorts.h"
#include "cuda/cuda_sorts.h"

// #define PERFORMANCE_DEBUG // if defined, prints the array after sorting it

constexpr char TEST_PATH[] = "tests/simpleint_2^20(-500000000_500000000)"; // test file path
constexpr int NRUNS = 10; // number of runs

int* iArray = nullptr;
float* fArray = nullptr;
int* itArray = nullptr;
float* ftArray = nullptr;

int size = 0; 
int useFloat = false;

template<typename Num>
void copyArr(Num* des, Num* src, int N) {
    for (int i = 0; i < N; ++i)
        des[i] = src[i];
}

void loadArray();
void finish();

int main() {
    loadArray();

    #ifdef PERFORMANCE_DEBUG
        printf("array :\n");
        for (int i = 0; i < size; ++i) {
            if (useFloat)
                printf("%f ", fArray[i]);
            else
                printf("%i ", iArray[i]);
        }
        printf("\n\n");
    #endif

    double elapsedTime = 0;
    double pre = 0;
    int min = -500000000, max = 500000000;
    printf("sorting %s array of size %d in range (%d, %d)...\n", useFloat ? "Float" : "Int", size, min, max);
    for (int i = 0; i < NRUNS; ++i) {
        if (useFloat) {
            // copyArr(ftArray, fArray, size);
            // cuda_par::mergeSort(ftArray, size, elapsedTime);

            // double t = omp_get_wtime();
            // std::sort(ftArray, ftArray + size);
            // elapsedTime += omp_get_wtime() - t;
        }
        else {
            copyArr(itArray, iArray, size);
            cuda_par::countingSort(itArray, size, min, max, elapsedTime);
            // cuda_par::countingSort(itArray, size, min, max, elapsedTime);

            // double t = omp_get_wtime();
            // std::sort(itArray, itArray + size);
            // elapsedTime += omp_get_wtime() - t;
        }
        printf("\tRun number %d completed in %f seconds.\n", i + 1, elapsedTime - pre);
        pre = elapsedTime;
    }
    printf("\nsorting completed in %f seconds in average.\n", elapsedTime / NRUNS);

    #ifdef PERFORMANCE_DEBUG
        printf("\nsorted array :\n");
        for (int i = 0; i < size; ++i) {
            if (useFloat)
                printf("%f ", fArray[i]);
            else
                printf("%i ", iArray[i]);
        }
        printf("\n");
    #endif

    finish();
}

void loadArray() {
    printf("Opening the file \'%s\' for reading...\n", TEST_PATH);
    FILE* fp = fopen(TEST_PATH, "r");
    
    fscanf(fp, "%d", &useFloat);
    fscanf(fp, "%d", &size);

    printf("File opend succesfully.\n \tuseFloat: %s\n\tsize: %d\n", useFloat ? "True" : "False", size);

    printf("loading the array...\n");
    if (useFloat) {
        fArray = new float[size];
        ftArray = new float[size];
    }
    else {
        iArray = new int[size];
        itArray = new int[size];
    }

	for (int i = 0; i < size; ++i)
	{
        if (useFloat) {
            fscanf(fp, "%f", fArray + i);
            ftArray[i] = fArray[i];
        }
        else {
            fscanf(fp, "%d", iArray + i);
            itArray[i] = iArray[i];
        }
	}

	fclose(fp);

	printf("\n#### Array loaded succesfully! ####\n\n");
}

void finish() {
    printf("\nfreeing memory...\n");

    if (iArray != nullptr)
        delete[] iArray;
    if (fArray != nullptr)
        delete[] fArray;
    if (itArray != nullptr)
        delete[] itArray;
    if (ftArray != nullptr)
        delete[] ftArray;
}